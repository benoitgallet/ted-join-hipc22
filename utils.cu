#include <stdio.h>

#include "utils.h"


void cudaErrCheck_(hipError_t errCode, const char* file, int line)
{
    if (errCode != hipSuccess)
    {
        fprintf(stderr, "[Error] ~ In %s (line %d): %s.\n", file, line, hipGetErrorString(errCode));
    }
}
